#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "timer.hpp"

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

//
// CUDA KERNELS
//
__global__ void cuda_step1()
{

}

__global__ void cuda_step2()
{
    
}

__global__ void cuda_step3()
{
    
}

void run_simulation_gpu(const SimInput_t *input, SimOutput_t *output)
{
    cuda_step1<<<,>>>();
    //memory management
    cuda_step2<<<,>>>();
    //memory management
    cuda_step3<<<,>>>();
    //memory management
}

//
// Data container for simulation input
//
typedef struct
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    size_t population_size; // Number of people to simulate
    double *rand_array; // Random numbers

    //// Configuration
    int mask_threshold;      // Number of cases required for masks
    int lockdown_threshold;  // Number of cases required for lockdown
    int infection_delay;     // Number of days before an infected person can pass on the disease
    int infection_days;      // Number of days an infected person can pass on the disease
    int starting_infections; // Number of infected people at the start of the year
    int immunity_duration;   // Number of days a recovered person is immune

    // for each day:
    int *contacts_per_day;            // number of other persons met each day to whom the disease may be passed on
    double *transmission_probability; // how likely it is to pass on the infection to another person

    ////////////////////
    // GPU ressources //
    ////////////////////

    double *rand_array_dev; // Random numbers
    int *contacts_per_day_dev;   
    double *transmission_probability_dev;

} SimInput_t;

void init_input(SimInput_t *input)
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    input->population_size = 8916845;               // Austria's population in 2020 according to Statistik Austria

    num_rands = input->population_size * 2 * 365;
    input->rand_array = (double *)malloc(sizeof(double) * num_rands); // fill random number array
    srand(0);                                                         // initialize random seed
    for (int i = 0; i < num_rands; i++)
    {
        input->rand_array[i] = ((double)rand()) / (double)RAND_MAX; // random number between 0 and 1
    }

    input->mask_threshold = 5000;
    input->lockdown_threshold = 50000;
    input->infection_delay = 5; // 5 to 6 days incubation period (average) according to WHO
    input->infection_days = 3;  // assume three days of passing on the disease
    input->starting_infections = 10;
    input->immunity_duration = 180; // half a year of immunity

    input->contacts_per_day = (int *)malloc(sizeof(int) * 365);
    input->transmission_probability = (double *)malloc(sizeof(double) * 365);
    for (int day = 0; day < 365; ++day)
    {
        input->contacts_per_day[day] = 6;                                                 // arbitrary assumption of six possible transmission contacts per person per day, all year
        input->transmission_probability[day] = 0.2 + 0.1 * cos((day / 365.0) * 2 * M_PI); // higher transmission in winter, lower transmission during summer
    }

    ////////////////////
    // GPU ressources //
    ////////////////////
    
    //step1

    //step2
    hipMalloc(&input->contacts_per_day_dev, sizeof(int) * 365);
    hipMemcpy(input->contacts_per_day_dev, input->contacts_per_day, sizeof(int) * 365, hipMemcpyHostToDevice);

    hipMalloc(&input->transmission_probability_dev, sizeof(double) * 365);
    hipMemcpy(input->transmission_probability_dev, input->transmission_probability, sizeof(double) * 365, hipMemcpyHostToDevice);

    //step3
    hipMalloc(&input->rand_array_dev, sizeof(double) * (num_rands));
    hipMemcpy(input->rand_array_dev, input->rand_array, sizeof(double) * (num_rands), hipMemcpyHostToDevice);
}

typedef struct
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    // for each day:
    int *active_infections; // number of active infected on that day (including incubation period)
    int *lockdown;          // 0 if no lockdown on that day, 1 if lockdown

    // for each person:
    int *is_infected; // 0 if healthy, 1 if currently infected
    int *infected_on; // day of infection. negative if not yet infected. January 1 is Day 0.

    ////////////////////
    // GPU ressources //
    ////////////////////

    // for each day:
    int *active_infections_dev; 
    int *lockdown_dev;          

    // step 1& step 3 : for each person 
    int *is_infected_dev; 
    int *infected_on_dev;

} SimOutput_t;

//
// Initializes the output data structure (values to zero, allocate arrays)
//
void init_output(SimOutput_t *output, int population_size)
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    output->active_infections = (int *)malloc(sizeof(int) * 365);
    output->lockdown = (int *)malloc(sizeof(int) * 365);
    for (int day = 0; day < 365; ++day)
    {
        output->active_infections[day] = 0;
        output->lockdown[day] = 0;
    }

    output->is_infected = (int *)malloc(sizeof(int) * population_size);
    output->infected_on = (int *)malloc(sizeof(int) * population_size);

    for (int i = 0; i < population_size; ++i)
    {
        output->is_infected[i] = 0;
        output->infected_on[i] = 0;
    }
    
    ////////////////////
    // GPU ressources //
    ////////////////////
    
    //step2
    hipMalloc(&output->active_infections_dev, sizeof(int) * 365);
    hipMalloc(&output->lockdown_dev, sizeof(int) * 365);
    hipMemcpy(output->active_infections_dev, output->active_infections, sizeof(int) * 365, hipMemcpyHostToDevice);
    hipMemcpy(output->lockdown_dev, output->lockdown, sizeof(int) * 365, hipMemcpyHostToDevice);

    //step1 & step3
    hipMalloc(&output->is_infected_dev, sizeof(int) * population_size);
    hipMalloc(&output->infected_on_dev, sizeof(int) * population_size);
    hipMemcpy(output->is_infected_dev, output->is_infected, sizeof(int) * population_size, hipMemcpyHostToDevice);
    hipMemcpy(output->infected_on_dev, output->infected_on, sizeof(int) * population_size, hipMemcpyHostToDevice);
}

void destruction(SimInput_t input, SimOutput_t output)
{
    // input stuff
    free(input->rand_array);
    free(input->contacts_per_day);
    free(input->transmission_probability);

    hipFree(input->rand_array_dev);
    hipFree(input->contacts_per_day_dev);
    hipFree(input->transmission_probability_dev);

    //output stuff
    free(output->active_infections);
    free(output->lockdown);
    free(output->is_infected);
    free(output->infected_on);

    hipFree(output->active_infections_dev);
    hipFree(output->lockdown_dev);
    hipFree(output->is_infected_dev);
    hipFree(output->infected_on_dev);
}
void run_simulation(const SimInput_t *input, SimOutput_t *output)
{
    //
    // Init data. For simplicity we set the first few people to 'infected'
    //
    for (int i = 0; i < input->population_size; ++i)
    {
        output->is_infected[i] = (i < input->starting_infections) ? 1 : 0;
        output->infected_on[i] = (i < input->starting_infections) ? 0 : -1; // infected on January 1
    }

    //
    // Run simulation
    //
    for (int day = 0; day < 365; ++day) // loop over all days of the year
    {
        //
        // Step 1: determine number of infections and recoveries
        //
        int num_infected_current = 0;
        int num_recovered_current = 0;
        for (int i = 0; i < input->population_size; ++i)
        {

            if (output->is_infected[i] > 0) // if person i is infected
            {
                if (output->infected_on[i] > day - input->infection_delay - input->infection_days && output->infected_on[i] <= day - input->infection_delay) // currently infected and incubation period over
                    num_infected_current += 1;
                else if (output->infected_on[i] < day - input->infection_delay - input->infection_days) // both incubation and infectionous time are over
                    num_recovered_current += 1;
            }
        }

        output->active_infections[day] = num_infected_current;
        if (num_infected_current > input->lockdown_threshold)
        {
            output->lockdown[day] = 1;
        }
        if (day > 0 && output->lockdown[day - 1] == 1)
        { // end lockdown if number of infections has reduced significantly
            output->lockdown[day] = (num_infected_current < input->lockdown_threshold / 3) ? 0 : 1;
        }
        char lockdown[] = " [LOCKDOWN]";
        char normal[] = "";
        printf("Day %d%s: %d active, %d recovered\n", day, output->lockdown[day] ? lockdown : normal, num_infected_current, num_recovered_current);

        //
        // Step 2: determine today's transmission probability and contacts based on pandemic situation
        //
        double contacts_today = input->contacts_per_day[day];
        double transmission_probability_today = input->transmission_probability[day];
        if (num_infected_current > input->mask_threshold)
        { // transmission is reduced with masks. Arbitrary factor: 2
            transmission_probability_today /= 2.0;
        }
        if (output->lockdown[day])
        { // contacts are significantly reduced in lockdown. Arbitrary factor: 4
            contacts_today /= 4;
        }

        //
        // Step 3: pass on infections within population
        //
        for (int i = 0; i < input->population_size; ++i) // loop over population
        {
            if (output->is_infected[i] > 0 && output->infected_on[i] > day - input->infection_delay - input->infection_days // currently infected
                && output->infected_on[i] <= day - input->infection_delay)                                                  // already infectious
            {
                // pass on infection to other persons with transmission probability
                for (int j = 0; j < contacts_today; ++j)
                {
                    double r = ((double)rand()) / (double)RAND_MAX; // random number between 0 and 1
                    if (r < transmission_probability_today)
                    {
                        r = ((double)rand()) / (double)RAND_MAX; // new random number to determine a random other person to transmit the virus to
                        int other_person = r * input->population_size;
                        if (output->is_infected[other_person] == 0                                 // other person is not infected
                            || output->infected_on[other_person] < day - input->immunity_duration) // other person has no more immunity
                        {
                            output->is_infected[other_person] = 1;
                            output->infected_on[other_person] = day;
                        }
                    }

                } // for contacts_per_day
            }     // if currently infected
        }         // for i

    } // for day
}

int main(int argc, char **argv)
{

    SimInput_t input;
    SimOutput_t output;

    init_input(&input);
    init_output(&output, input.population_size);

    Timer timer;

    srand(0); // initialize random seed for deterministic output
    timer.reset();
    run_simulation(&input, &output);
    printf("Simulation time: %g\n", timer.get());

    destruction(&input, &output);

    return EXIT_SUCCESS;
}