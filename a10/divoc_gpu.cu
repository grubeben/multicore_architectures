#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "timer.hpp"

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// KERNEL RUN PARAMETERS
#define BLOCK_NUMBER 256
#define THREADS_PER_BLOCK 256


////////////////////////////////////////////////////////////////////////////////////////////////
// CUDA KERNELS ////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////


// Step 1 2 3 in one kernel: 
// 1: determine number of infections and recoveries;
// 2: determine today's transmission probability and contacts based on pandemic situation;
// 3: pass on infections within population;
__global__ void cuda_step123(int day, const SimInput_t *input, SimOutput_t *output)
{
    // STEP1
    // every thread counts the infected/recovered it handles (this is inspired by the dot product)
    __shared__ int num_infected_current_shared[256];
    __shared__ int num_recovered_current_shared[256];

    // let every thread deal with one person at a time
    int num_infected_current_local = 0;
    int num_recovered_current_local = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input->population_size; i += blockDim.x * gridDim.x)
    {
        if (output->is_infected_dev[i] > 0) // if person i is infected
        {
            if (output->infected_on_dev[i] > day - input->infection_delay - input->infection_days && output->infected_on_dev[i] <= day - input->infection_delay) // currently infected and incubation period over
                num_infected_current += 1;
            else if (output->infected_on_dev[i] < day - input->infection_delay - input->infection_days) // both incubation and infectionous time are over
                num_recovered_current += 1;
        }
    }
    // we need to sync threads here and reduce
    num_infected_current_shared[threadIdx.x] = num_infected_current_local;
    num_recovered_current_shared[threadIdx.x] = num_recovered_current_local;
    for (int k = blockDim.x / 2; k > 0; k /= 2)
    {
        __syncthreads();
        if (threadIdx.x < k)
        {
            num_infected_current_shared[threadIdx.x] += num_infected_current_shared[threadIdx.x + k];
            num_recovered_current_shared[threadIdx.x] += num_recovered_current_shared[threadIdx.x + k];
        }
    }
    // after stride thread 0 holds block_sums, it will now AtomicAdd them to the ouput-> GPU arrays
    if (threadIdx.x == 0)
    {
        atomicAdd(output->active_infections_dev[day], num_infected_current_shared[0]);
    }
    // care for non-parallelizable stuff with only one thread
    if (blockIdx.x * blockDim.x + threadIdx.x==0)
    {
        if (day > 0 && output->lockdown_dev[day - 1] == 1)
        { // end lockdown if number of infections has reduced significantly
            output->lockdown_dev[day] = (active_infections_dev[day] < input->lockdown_threshold / 3) ? 0 : 1;
        }
        // daily announcement
        char lockdown[] = " [LOCKDOWN]";
        char normal[] = "";
        printf("Day %d%s: %d active, %d recovered\n", day, output->lockdown_dev[day] ? lockdown : normal, active_infections_dev[day], num_recovered_current_shared[0]);
    

        //STEP2 
        if (active_infections_dev[day] > input->mask_threshold) { // transmission is reduced with masks. Arbitrary factor: 2
        input->transmission_probability_dev[day] /= 2.0;
        }
        if (output->lockdown_dev[day]) { // contacts are significantly reduced in lockdown. Arbitrary factor: 4
        input->contacts_per_day_dev[day] /= 4;
        }
    }

    //not sure if necessary, but I want to ensure every thread grabs the manipulated values (STEP2)
    __syncthreads();

    double contacts_today = input->contacts_per_day[day];
    double transmission_probability_today = input->transmission_probability[day];

    //STEP 3 - we back in parallel mode
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input->population_size; i += blockDim.x * gridDim.x)
    {
        if (output->is_infected_dev[i] > 0
          && output->infected_on_dev[i] >  day - input->infection_delay - input->infection_days  // currently infected
          && output->infected_on_dev[i] <= day - input->infection_delay)                         // already infectious
        {
            // pass on infection to other persons with transmission probability
            for (int j=0; j<contacts_today; ++j)
            {
                double r = input->rand_array_dev[i];  // random number between 0 and 1
                if (r < transmission_probability_today)
                {
                    r = input->rand_array_dev[2*i];       // new random number to determine a random other person to transmit the virus to
                    int other_person = r * input->population_size;

                    // SHOULD THIS BE A SEQUENTIAL SECTION?
                    ////////////////////////////////////////////////////////////////////////
                    if (output->is_infected_dev[other_person] == 0     // other person is not infected
                        || output->infected_on_dev[other_person] < day - input->immunity_duration)  // other person has no more immunity
                    {
                        output->is_infected_dev[other_person] = 1;
                        output->infected_on_dev[other_person] = day;
                    }
                    ///////////////////////////////////////////////////////////////////////
                }
            }
        }
    }
}

// wrap kernels; init_input and init_output must be called prior because they fill dev arrays
void run_simulation_gpu(const SimInput_t *input, SimOutput_t *output)
{
    for (int day = 0; day < 365; ++day) // loop over all days of the year
    {
        cuda_step123<<<BLOCK_NUMBER, THREADS_PER_BLOCK>>>(day,input,output);

        // hand back current_infections_dev back to CPU as task demands?
        // but we can simply print within kernel, why create overhead?
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Data container //////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////


//INPUT DATA
typedef struct
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    size_t population_size; // Number of people to simulate
    double *rand_array;     // Random numbers

    //// Configuration
    int mask_threshold;      // Number of cases required for masks
    int lockdown_threshold;  // Number of cases required for lockdown
    int infection_delay;     // Number of days before an infected person can pass on the disease
    int infection_days;      // Number of days an infected person can pass on the disease
    int starting_infections; // Number of infected people at the start of the year
    int immunity_duration;   // Number of days a recovered person is immune

    // for each day:
    int *contacts_per_day;            // number of other persons met each day to whom the disease may be passed on
    double *transmission_probability; // how likely it is to pass on the infection to another person

    ////////////////////
    // GPU ressources //
    ////////////////////

    double *rand_array_dev; // Random numbers
    int *contacts_per_day_dev;
    double *transmission_probability_dev;
} SimInput_t;

//OUTPUT DATA
typedef struct
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    // for each day:
    int *active_infections; // number of active infected on that day (including incubation period)
    int *lockdown;          // 0 if no lockdown on that day, 1 if lockdown

    // for each person:
    int *is_infected; // 0 if healthy, 1 if currently infected
    int *infected_on; // day of infection. negative if not yet infected. January 1 is Day 0.

    ////////////////////
    // GPU ressources //
    ////////////////////

    // for each day:
    int *active_infections_dev;
    int *active_recovered_dev; //number of recovered people for each day
    int *lockdown_dev;

    // step 1& step 3 : for each person
    int *is_infected_dev;
    int *infected_on_dev;
} SimOutput_t;

// FREE STUFF
// there must be a better way of doing this (as part of the struct?)
void destruction(SimInput_t input, SimOutput_t output)
{
    // input struct
    free(input->rand_array);
    free(input->contacts_per_day);
    free(input->transmission_probability);

    hipFree(input->rand_array_dev);
    hipFree(input->contacts_per_day_dev);
    hipFree(input->transmission_probability_dev);

    // output struct
    free(output->active_infections);
    free(output->lockdown);
    free(output->is_infected);
    free(output->infected_on);

    hipFree(output->active_infections_dev);
    hipFree(output->lockdown_dev);
    hipFree(output->is_infected_dev);
    hipFree(output->infected_on_dev);
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Data Initialization /////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////

// INPUT DATA
void init_input(SimInput_t *input)
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    input->population_size = 8916845; // Austria's population in 2020 according to Statistik Austria

    //RAND NUMBERS VERSION 1: generate rand number array on CPU and copy to GPU
    num_rands = input->population_size * 2 * 365;
    input->rand_array = (double *)malloc(sizeof(double) * num_rands); // fill random number array
    srand(0);                                                         // initialize random seed
    for (int i = 0; i < num_rands; i++)
    {
        input->rand_array[i] = ((double)rand()) / (double)RAND_MAX; // random number between 0 and 1
    }

    input->mask_threshold = 5000;
    input->lockdown_threshold = 50000;
    input->infection_delay = 5; // 5 to 6 days incubation period (average) according to WHO
    input->infection_days = 3;  // assume three days of passing on the disease
    input->starting_infections = 10;
    input->immunity_duration = 180; // half a year of immunity

    input->contacts_per_day = (int *)malloc(sizeof(int) * 365);
    input->transmission_probability = (double *)malloc(sizeof(double) * 365);
    for (int day = 0; day < 365; ++day)
    {
        input->contacts_per_day[day] = 6;                                                 // arbitrary assumption of six possible transmission contacts per person per day, all year
        input->transmission_probability[day] = 0.2 + 0.1 * cos((day / 365.0) * 2 * M_PI); // higher transmission in winter, lower transmission during summer
    }

    ////////////////////
    // GPU ressources //
    ////////////////////

    // step2
    hipMalloc(&input->contacts_per_day_dev, sizeof(int) * 365);
    hipMemcpy(input->contacts_per_day_dev, input->contacts_per_day, sizeof(int) * 365, hipMemcpyHostToDevice);

    hipMalloc(&input->transmission_probability_dev, sizeof(double) * 365);
    hipMemcpy(input->transmission_probability_dev, input->transmission_probability, sizeof(double) * 365, hipMemcpyHostToDevice);

    // RAND NUMBERS VERSION 1
    hipMalloc(&input->rand_array_dev, sizeof(double) * (num_rands));
    hipMemcpy(input->rand_array_dev, input->rand_array, sizeof(double) * (num_rands), hipMemcpyHostToDevice);

    // RAND NUMBERS VERSION 2
    
}

// OUTPUT DATA
void init_output(SimOutput_t *output, int population_size)
{
    ////////////////////
    // CPU ressources //
    ////////////////////

    output->active_infections = (int *)malloc(sizeof(int) * 365);
    output->lockdown = (int *)malloc(sizeof(int) * 365);
    for (int day = 0; day < 365; ++day)
    {
        output->active_infections[day] = 0;
        output->lockdown[day] = 0;
    }

    output->is_infected = (int *)malloc(sizeof(int) * population_size);
    output->infected_on = (int *)malloc(sizeof(int) * population_size);

    for (int i = 0; i < population_size; ++i)
    {
        output->is_infected[i] = 0;
        output->infected_on[i] = 0;
    }

    ////////////////////
    // GPU ressources //
    ////////////////////

    // step2
    hipMalloc(&output->active_infections_dev, sizeof(int) * 365);
    hipMalloc(&output->active_recovered_dev, sizeof(int) * 365);
    hipMalloc(&output->lockdown_dev, sizeof(int) * 365);
    hipMemcpy(output->active_infections_dev, output->active_infections, sizeof(int) * 365, hipMemcpyHostToDevice);
    hipMemcpy(output->active_recovered_dev;, output->is_infected, sizeof(int) * 365, hipMemcpyHostToDevice); // use is_infected to initialize with zeros
    hipMemcpy(output->lockdown_dev, output->lockdown, sizeof(int) * 365, hipMemcpyHostToDevice);
    
    // step1 & step3
    hipMalloc(&output->is_infected_dev, sizeof(int) * population_size);
    hipMalloc(&output->infected_on_dev, sizeof(int) * population_size);
    hipMemcpy(output->is_infected_dev, output->is_infected, sizeof(int) * population_size, hipMemcpyHostToDevice);
    hipMemcpy(output->infected_on_dev, output->infected_on, sizeof(int) * population_size, hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////////////////////
// MAIN ////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{

    SimInput_t input;
    SimOutput_t output;

    init_input(&input);
    init_output(&output, input.population_size);

    Timer timer;

    timer.reset();
    run_simulation_gpu(&input, &output);
    printf("Simulation time: %g\n", timer.get());

    destruction(&input, &output);

    return EXIT_SUCCESS;
}