#include "hip/hip_runtime.h"
#include "poisson2d.hpp"
#include "timer.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include "cuda_errchk.hpp"

// y = A * x
__global__ void cuda_csr_matvec_product(int NN, int *csr_rowoffsets,
                                        int *csr_colindices, double *csr_values,
                                        double *x, double *y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < NN; i += blockDim.x * gridDim.x)
    {
        double sum = 0;
        for (int k = csr_rowoffsets[i]; k < csr_rowoffsets[i + 1]; k++)
        {
            sum += csr_values[k] * x[csr_colindices[k]];
        }
        y[i] = sum;
    }
}

// x <- x + alpha * y
__global__ void cuda_vecadd(int NN, double *x, double *y, double alpha)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < NN; i += blockDim.x * gridDim.x)
        x[i] += alpha * y[i];
}

// x <- y + alpha * x
__global__ void cuda_vecadd2(int NN, double *x, double *y, double alpha)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < NN; i += blockDim.x * gridDim.x)
        x[i] = y[i] + alpha * x[i];
}

// result = (x, y)
__global__ void cuda_dot_product(int NN, double *x, double *y, double *result)
{
    __shared__ double shared_mem[512];

    double dot = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < NN; i += blockDim.x * gridDim.x)
    {
        dot += x[i] * y[i];
    }

    shared_mem[threadIdx.x] = dot;
    for (int k = blockDim.x / 2; k > 0; k /= 2)
    {
        __syncthreads();
        if (threadIdx.x < k)
        {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + k];
        }
    }

    if (threadIdx.x == 0)
        atomicAdd(result, shared_mem[0]);
}

__global__ void count_nnz(int *nn_counts, int N, int M)
// each row describes one node, hence simply by the position of the row in the matrix,
// we can deduce how many entries are expected to be populated
{
    for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < N * M; row += gridDim.x * blockDim.x)
    {
        int nnz_for_this_node = 1;
        int i = row / N;
        int j = row % N;

        if (i > 0)
            nnz_for_this_node += 1;
        if (j > 0)
            nnz_for_this_node += 1;
        if (i < N - 1)
            nnz_for_this_node += 1;
        if (j < M - 1)
            nnz_for_this_node += 1;

        nn_counts[row] = nnz_for_this_node;
    }
}

__global__ void populate_matrix(int *row_offsets, double *values, int *col_indices, int N, int M)
{
    for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < N * M; row += gridDim.x * blockDim.x)
    {
        int i = row / N;
        int j = row % N;
        int this_row_offset = row_offsets[row];

        // diagonal entry
        col_indices[this_row_offset] = i * N + j;
        values[this_row_offset] = 4;
        this_row_offset += 1;

        // upper neighbor
        if (i > 0)
        {
            // col_indices[this_row_offset] = (i-1)* N+j;
            col_indices[this_row_offset] = (i - 1) + N * j;
            values[this_row_offset] = -1;
            this_row_offset += 1;
        }

        // left neighbor
        if (j > 0)
        {
            // col_indices[this_row_offset] = i* N +(j-1);
            col_indices[this_row_offset] = i + N * (j - 1);
            values[this_row_offset] = -1;
            this_row_offset += 1;
        }

        // lower neighbor
        if (i < N - 1)
        {
            col_indices[this_row_offset] = (i + 1) + N * j;
            // col_indices[this_row_offset] = (i+1)* N +j;
            values[this_row_offset] = -1;
            this_row_offset += 1;
        }

        // right neighbour
        if (j < M - 1)
        {
            // col_indices[this_row_offset] = i*N +(j+1);
            col_indices[this_row_offset] = i + N * (j + 1);
            values[this_row_offset] = -1;
            this_row_offset += 1;
        }
    }
}

__global__ void scan_kernel_1(int const *X, int *Y, int N, int *carries)
{
    __shared__ int shared_buffer[256];
    int my_value;

    unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
    unsigned int block_start = work_per_thread * blockDim.x * blockIdx.x;
    unsigned int block_stop = work_per_thread * blockDim.x * (blockIdx.x + 1);
    unsigned int block_offset = 0;

    // run scan on each section
    for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
    {
        // load data:
        my_value = (i < N) ? X[i] : 0; // conditional operator if(i<N){my_value=X[i];}else{my_value=0}

        // inclusive scan in shared buffer:
        for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
        {
            __syncthreads();
            shared_buffer[threadIdx.x] = my_value;
            __syncthreads();
            if (threadIdx.x >= stride)
                my_value += shared_buffer[threadIdx.x - stride];
        }
        __syncthreads();
        shared_buffer[threadIdx.x] = my_value;
        __syncthreads();

        // exclusive scan requires us to write a zero value at the beginning of each block
        my_value = (threadIdx.x > 0) ? shared_buffer[threadIdx.x - 1] : 0;

        // write to output array
        if (i < N)
            Y[i] = block_offset + my_value;

        block_offset += shared_buffer[blockDim.x - 1];
    }

    // write carry:
    if (threadIdx.x == 0)
        carries[blockIdx.x] = block_offset;
}

// exclusive-scan of carries
__global__ void scan_kernel_2(int *carries)
{
    __shared__ int shared_buffer[256];

    // load data:
    int my_carry = carries[threadIdx.x];

    // exclusive scan in shared buffer:

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();
        shared_buffer[threadIdx.x] = my_carry;
        __syncthreads();
        if (threadIdx.x >= stride)
            my_carry += shared_buffer[threadIdx.x - stride];
    }
    __syncthreads();
    shared_buffer[threadIdx.x] = my_carry;
    __syncthreads();

    // write to output array
    carries[threadIdx.x] = (threadIdx.x > 0) ? shared_buffer[threadIdx.x - 1] : 0;
}

__global__ void scan_kernel_3(int *Y, int N,
                              int const *carries)
{
    unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
    unsigned int block_start = work_per_thread * blockDim.x * blockIdx.x;
    unsigned int block_stop = work_per_thread * blockDim.x * (blockIdx.x + 1);

    __shared__ int shared_offset;

    if (threadIdx.x == 0)
        shared_offset = carries[blockIdx.x];

    __syncthreads();

    // add offset to each element in the block:
    for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
        if (i < N)
            Y[i] += shared_offset;
}

void exclusive_scan(int const *input,
                    int *output, int N)
{
    int num_blocks = 256;
    int threads_per_block = 256;

    int *carries;
    hipMalloc(&carries, sizeof(int) * num_blocks);

    // First step: Scan within each thread group and write carries
    scan_kernel_1<<<num_blocks, threads_per_block>>>(input, output, N, carries);

    // Second step: Compute offset for each thread group (exclusive scan for each thread group)
    scan_kernel_2<<<1, num_blocks>>>(carries);

    // Third step: Offset each thread group accordingly
    scan_kernel_3<<<num_blocks, threads_per_block>>>(output, N, carries);

    hipFree(carries);
}

// compute averages
float med(std::vector<float> log_vec)
{
    return log_vec[log_vec.size() / 2];
}

/** Implementation of the conjugate gradient algorithm.
 *
 *  The control flow is handled by the CPU.
 *  Only the individual operations (vector updates, dot products, sparse
 * matrix-vector product) are transferred to CUDA kernels.
 *
 *  The temporary arrays p, r, and Ap need to be allocated on the GPU for use
 * with CUDA. Modify as you see fit.
 */
void conjugate_gradient(int NN, // number of unknows
                        int *csr_rowoffsets, int *csr_colindices,
                        double *csr_values, double *rhs, double *solution)
//, double *init_guess)   // feel free to add a nonzero initial guess as needed
{
    // initialize timer
    Timer timer;

    // clear solution vector (it may contain garbage values):
    std::fill(solution, solution + NN, 0);

    // initialize work vectors:
    double alpha, beta;
    double *cuda_solution, *cuda_p, *cuda_r, *cuda_Ap, *cuda_scalar;
    hipMalloc(&cuda_p, sizeof(double) * NN);
    hipMalloc(&cuda_r, sizeof(double) * NN);
    hipMalloc(&cuda_Ap, sizeof(double) * NN);
    hipMalloc(&cuda_solution, sizeof(double) * NN);
    hipMalloc(&cuda_scalar, sizeof(double));

    hipMemcpy(cuda_p, rhs, sizeof(double) * NN, hipMemcpyHostToDevice);
    hipMemcpy(cuda_r, rhs, sizeof(double) * NN, hipMemcpyHostToDevice);
    hipMemcpy(cuda_solution, solution, sizeof(double) * NN, hipMemcpyHostToDevice);

    const double zero = 0;
    double residual_norm_squared = 0;
    hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
    cuda_dot_product<<<512, 512>>>(NN, cuda_r, cuda_r, cuda_scalar);
    hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);

    double initial_residual_squared = residual_norm_squared;

    int iters = 0;
    hipDeviceSynchronize();
    timer.reset();
    while (1)
    {

        // line 4: A*p:
        cuda_csr_matvec_product<<<512, 512>>>(NN, csr_rowoffsets, csr_colindices, csr_values, cuda_p, cuda_Ap);

        // lines 5,6:
        hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
        cuda_dot_product<<<512, 512>>>(NN, cuda_p, cuda_Ap, cuda_scalar);
        hipMemcpy(&alpha, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);
        alpha = residual_norm_squared / alpha;

        // line 7:
        cuda_vecadd<<<512, 512>>>(NN, cuda_solution, cuda_p, alpha);

        // line 8:
        cuda_vecadd<<<512, 512>>>(NN, cuda_r, cuda_Ap, -alpha);

        // line 9:
        beta = residual_norm_squared;
        hipMemcpy(cuda_scalar, &zero, sizeof(double), hipMemcpyHostToDevice);
        cuda_dot_product<<<512, 512>>>(NN, cuda_r, cuda_r, cuda_scalar);
        hipMemcpy(&residual_norm_squared, cuda_scalar, sizeof(double), hipMemcpyDeviceToHost);

        // line 10:
        if (std::sqrt(residual_norm_squared / initial_residual_squared) < 1e-6)
        {
            break;
        }

        // line 11:
        beta = residual_norm_squared / beta;

        // line 12:
        cuda_vecadd2<<<512, 512>>>(NN, cuda_p, cuda_r, beta);

        if (iters > 10000)
            break; // solver didn't converge
        ++iters;
    }
    hipMemcpy(solution, cuda_solution, sizeof(double) * NN, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    std::cout << "Time elapsed: " << timer.get() << " (" << timer.get() / iters << " per iteration)" << std::endl;

    if (iters > 1000)
        std::cout << "Conjugate Gradient did NOT converge within 10000 iterations"
                  << std::endl;
    else
        std::cout << "Conjugate Gradient converged in " << iters << " iterations."
                  << std::endl;

    hipFree(cuda_p);
    hipFree(cuda_r);
    hipFree(cuda_Ap);
    hipFree(cuda_solution);
    hipFree(cuda_scalar);
}

/** Solve a system with `N * N` unknowns
 */
void solve_system(int N)
{
    int NN = N * N; // number of unknows to solve for

    std::cout << "Solving Ax=b with " << NN << " unknowns." << std::endl;

    // Allocation sizes
    int n_values = 5 * (N - 2) * (N - 2) + 4 * 4 * (N - 2) + 4 * 3; // 5*N*N is definitely sufficient, can we go exact? yes
    // int n_values = 5*NN;

    // Allocate host arrays
    int *csr_rowoffsets = (int *)malloc(sizeof(double) * (NN + 1)); // N*M nodes ==> N*M rows
    int *nn_counts = (int *)malloc(sizeof(int) * (NN));
    double *csr_values = (double *)malloc(sizeof(double) * n_values);
    int *csr_colindices = (int *)malloc(sizeof(double) * n_values);

    // Allocate CUDA-arrays
    int *cuda_csr_rowoffsets, *cuda_nn_counts, *cuda_csr_col_indices;
    double *cuda_csr_values;

    hipMalloc(&cuda_csr_rowoffsets, sizeof(double) * (NN + 1));
    hipMalloc(&cuda_nn_counts, sizeof(int) * (NN));
    hipMalloc(&cuda_csr_values, sizeof(double) * n_values);
    hipMalloc(&cuda_csr_col_indices, sizeof(double) * n_values);

    //
    // fill CSR matrix with values ==> PLUG SELFMADE ASSEMBLY IN HERE
    //

    // generate_fdm_laplace(N, csr_rowoffsets, csr_colindices,csr_values);
    count_nnz<<<256, 256>>>(cuda_nn_counts, N, N);                                                   // a)
    exclusive_scan(cuda_nn_counts, cuda_csr_rowoffsets, NN + 1);                                     // b)
    populate_matrix<<<256, 256>>>(cuda_csr_rowoffsets, cuda_csr_values, cuda_csr_col_indices, N, N); // c)

    //
    // Copy to host for rel resiudal computation
    //
    hipMemcpy(csr_rowoffsets, cuda_csr_rowoffsets, sizeof(double) * (NN + 1), hipMemcpyDeviceToHost);
    hipMemcpy(csr_colindices, cuda_csr_col_indices, sizeof(double) * n_values, hipMemcpyDeviceToHost);
    hipMemcpy(csr_values, cuda_csr_values, sizeof(double) * n_values, hipMemcpyDeviceToHost);

    //
    // Allocate solution vector and right hand side:
    //
    double *solution = (double *)malloc(sizeof(double) * NN);
    double *rhs = (double *)malloc(sizeof(double) * NN);
    std::fill(rhs, rhs + NN, 1);

    //
    // Call Conjugate Gradient implementation with GPU arrays
    //
    conjugate_gradient(NN, cuda_csr_rowoffsets, cuda_csr_col_indices, cuda_csr_values, rhs, solution);

    //
    // Check for convergence:
    //
    double residual_norm = relative_residual(NN, csr_rowoffsets, csr_colindices, csr_values, rhs, solution);
    std::cout << "Relative residual norm: " << residual_norm
              << " (should be smaller than 1e-6)" << std::endl;

    for (int i = 0; i <NN; i++) {std::cout <<solution[i] << std::endl;}
    hipFree(cuda_csr_rowoffsets);
    hipFree(cuda_csr_col_indices);
    hipFree(cuda_csr_values);
    free(solution);
    free(rhs);
    free(csr_rowoffsets);
    free(csr_colindices);
    free(csr_values);
}


int main()
{
    solve_system(10);
    return EXIT_SUCCESS;
}
